#include "hip/hip_runtime.h"
using namespace cooperative_groups;

__global__ 
void cooperative_kernel(int *dev_buf) {

	// it seems that CUDA9 do not support sm_21

	printf("cooperative_kernel\n");

	thread_group block = this_thread_block();
	thread_group tile32 = tiled_partition(block, 32);

	if (block.thread_rank() < 32) {
	    tile32.sync();
	}

	block.sync();

	dev_buf[0] = 2;
}

void test() {
	#define BUF_SIZE      100

	printf("test\n");

	int *host_buf = (int *)malloc(BUF_SIZE);
	int *dev_buf = NULL;
	CU_CHECK(hipMalloc((void **)&dev_buf, BUF_SIZE));
	CU_CHECK(hipMemset(dev_buf, 0, BUF_SIZE));

	dim3 dimGrid(1);
	dim3 dimBlock(1, 1, 1);

	cooperative_kernel<<<dimGrid, dimBlock>>>(dev_buf);
	CU_CHECK(hipDeviceSynchronize());

	host_buf[0] = 100;
	CU_CHECK(hipMemcpy(host_buf, dev_buf, BUF_SIZE, hipMemcpyDeviceToHost));

	printf("%d\n", host_buf[0]);
}






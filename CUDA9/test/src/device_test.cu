void test(){
	hipError_t err = hipSuccess;
	hipDeviceProp_t prop;
	int count = 0;

	int version;
	CU_CHECK(hipRuntimeGetVersion(&version));
	printf("hipRuntimeGetVersion %d\n", version);
	CU_CHECK(hipDriverGetVersion(&version));
	printf("hipDriverGetVersion %d\n", version);

	hipGetDeviceCount(&count);

	for (int i = 0; i < count; i++) {
		printf("Device %d\n", i);
		prop.totalGlobalMem = 0;
		err = hipGetDeviceProperties(&prop, i);
		CU_CHECK(err);

		printf("  name: %s\n", prop.name);
		printf("  major.minor: %d.%d\n", prop.major, prop.minor);
		printf("  clockRate: %d\n", prop.clockRate);
		printf("  deviceOverlap: %d\n", prop.deviceOverlap);
		printf("  kernelExecTimeoutEnabled: %d\n", prop.kernelExecTimeoutEnabled);
		printf("  integrated: %d\n", prop.integrated);
		printf("\n");
		printf("  totalGlobalMem: %lld\n", prop.totalGlobalMem);
		printf("  totalConstMem: %ld\n", prop.totalConstMem);
		printf("  memPitch: %ld\n", prop.memPitch);
		printf("  textureAlignment: %ld\n", prop.textureAlignment);
		printf("\n");
		printf("  multiProcessorCount: %d\n", prop.multiProcessorCount);
		printf("  sharedMemPerBlock: %ld\n", prop.sharedMemPerBlock);
		printf("  regsPerBlock: %d\n", prop.regsPerBlock);
		printf("  warpSize: %d\n", prop.warpSize);
		printf("  maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
		printf("  maxThreadsDim: %d  %d  %d\n", 
			prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("  maxGridSize: %d  %d  %d\n", 
			prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\n");
	}
}



